#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
    // Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  	// Avoid accessing out of bounds elements
    if (i < N)
    {
    	a[i] = i % 2;
  	}
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
    // Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  	// Avoid accessing out of bounds elements
    if (i < N) 
    {
    	a[i] = blockIdx.x;
  	}
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
    // Compute the global element index this thread should process
  	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  	// Avoid accessing out of bounds elements
    if (i < N) 
    {
    	a[i] = threadIdx.x;
    }
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    for (int i = 0; i < nDevices; ++i)
    {
        hipDeviceProp_t currentDeviceProperties;
        hipGetDeviceProperties(&currentDeviceProperties, i);

        printf("%s\n", currentDeviceProperties.name);
        printf("%d\n", currentDeviceProperties.totalGlobalMem);
        printf("%d\n", currentDeviceProperties.memoryClockRate);
        printf("%d\n", currentDeviceProperties.memoryBusWidth);
    }

    hipSetDevice(0);

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.

    int block_size = 4;
    int blocks_no = 4;
    int num_elements = block_size * blocks_no;
  	const int num_bytes = num_elements * sizeof(int);

    int *host_array_a = (int*)malloc(num_bytes);

    int *device_array_a;
    hipMalloc((void **) &device_array_a, num_bytes);

    // If any memory allocation failed, report an error message
    if (host_array_a == 0 || device_array_a == 0) 
    {
    	printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }
      
    fill_array_int(host_array_a, num_elements);
    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);

    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results

    // Launch the kernel
    kernel_parity_id<<<blocks_no, block_size>>>(device_array_a, num_elements);
    hipDeviceSynchronize();
  	hipMemcpy(host_array_a, device_array_a, num_bytes, hipMemcpyDeviceToHost);
    
    check_task_1(3, host_array_a);

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results

    // Explicatie: Sunt 4 block-uri de dim 4, deci va pune aceeasi valoare de 4 ori, apoi se va schimba
    kernel_block_id<<<blocks_no, block_size>>>(device_array_a, num_elements);
    hipDeviceSynchronize();
  	hipMemcpy(host_array_a, device_array_a, num_bytes, hipMemcpyDeviceToHost);

    check_task_1(4, host_array_a);

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results
    
    // Explicatie: Sunt 4 thread-uri per block, deci va pune 0 1 2 3 0 1 2 3...
    kernel_thread_id<<<blocks_no, block_size>>>(device_array_a, num_elements);
    hipDeviceSynchronize();
  	hipMemcpy(host_array_a, device_array_a, num_bytes, hipMemcpyDeviceToHost);

    check_task_1(5, host_array_a);

    // TODO 6: Free the memory
    free(host_array_a);
	hipFree(device_array_a);
    return 0;
}